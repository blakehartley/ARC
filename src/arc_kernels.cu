#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "./inc/chealpix.h"
#include "./inc/chealpix.cu"
#include "./inc/rates.cu"

__device__ float energy[FREQ_BIN_NUM] = {16.74, 24.65, 34.49, 52.06};

__device__ float gfn[FREQ_BIN_NUM] = {	0.277, 
										0.335, 
										0.2, 
										0.188};

inline __device__ void sigma(float sig[][FREQ_BIN_NUM])
{
	sig[0][0] = 3.61240988816e-18;	// 16.74 eV
	sig[0][1] = 1.23064754251e-18;	// 24.65 eV
	sig[0][2] = 4.70615962674e-19;	// 34.49 eV
	sig[0][3] = 1.4045184426e-19;	// 52.06 eV
	sig[1][0] = 0;	// 16.74 eV
	sig[1][1] = 7.78171958766e-18;	// 24.65 eV
	sig[1][2] = 4.20508180238e-18;	// 34.49 eV
	sig[1][3] = 1.91002975156e-18;	// 52.06 eV
}

__device__ float time(float redshift) {
	float h = 0.6711;
	float h0 = h*3.246753e-18;
	float omegam = 0.3;
	float yrtos = 3.15569e7;
	float time = 2.*powf((1. + redshift), -3. / 2.) / (3.*h0*powf(omegam, 0.5));
	time = time / (yrtos*1.e6);
	return time;
}

__device__ float redshift(float time) {
	float h = 0.6711;
	float h0 = h*3.246753e-18;
	float omegam = 0.3;
	float yrtos = 3.15569e7;
	time = time*yrtos*1.e6;
	float redshift = powf((3.*h0*powf(omegam, 0.5)*time / 2.), -2. / 3.) - 1.;
	return redshift;
}

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ void step_bdf(float* yout, float* y, float* dGdx, float n, float E, float dt)
{
	float xe = (1.0-Y_P)*(1.0-y[0])+0.25*Y_P*(1.0-y[1]);
	float ne = n*xe;
	float T  = E/((3./2.)*8.6173303e-5)/(1.0+xe);
	int CASE = 1;
	
	// Recombination rates (Black81)
	float al[2];
	al[0] = rec_HII(T, CASE);
	al[1] = rec_HeII(T, CASE);
	
	// Collision Excitation
	float col[2];
	col[0] = col_HI(T);
	col[1] = col_HeI(T);
	
	for(int nBin=0; nBin < SPECIES; nBin++)
	{
		// Find the ionization rate per neutral particle
		/*float gamDer;
		if(y[nBin] < 1.e-30)
			gamDer = 0.0;
		else
			gamDer = gam[nBin]/y[nBin];*/
		
		float source = al[nBin]*(1.0-y[nBin])*ne*dt + y[nBin];
		float sink = 1.0 + (dGdx[nBin] + col[nBin]*ne)*dt;
		
		yout[nBin] = source/sink;
		/*yout[nBin] = y[nBin];
		yout[nBin] += dt*(-gam[nBin] - col[nBin]*ne*y[nBin]);
		yout[nBin] += dt*al[nBin]*(1.0-y[nBin])*ne;*/
	}
}

__device__ float dnHIdt(float* y, float* dGdx, float n, float E)
{
	float xe = (1.0-Y_P)*(1.0-y[0])+0.25*Y_P*(1.0-y[1]);
	float ne = n*xe;
	float T  = E/((3./2.)*8.6173303e-5)/(1.0+xe);
	int CASE = 1;
	
	// Recombination rates (Black81)
	float al[2];
	al[0] = rec_HII(T, CASE);
	al[1] = rec_HeII(T, CASE);
	
	// Collision Excitation
	float col[2];
	col[0] = col_HI(T);
	col[1] = col_HeI(T);
	
	int nBin=0;
	
	float x = -dGdx[nBin]*y[nBin] - col[nBin]*ne*y[nBin] + al[nBin]*(1.0-y[nBin])*ne;
	return x;
}

__device__ float lambda(float E, float* y, float n, float a)
{
	int CASE = 1;
	
	float dEdt = 0.0;
	float xe = (1.0-Y_P)*(1.0-y[0])+0.25*Y_P*(1.0-y[1]);
	float ne = n*xe;
	float T  = E/((3./2.)*8.6173303e-5)/(1.0+xe);
	
	float xHI = (1-Y_P)*y[0];
	float xHII = (1-Y_P)*(1.0 - y[0]);
	float xHeI = 0.25*Y_P*y[1];
	float xHeII = 0.0;
	//float xHeIII = 0.0;
	
	float colH = col_cool_HI(T)*ne*xHI;
	float colHe = col_cool_HeI(T)*ne*xHeI;
	
	float recH =  rec_cool_HII(T, CASE)*ne*xHII;
	float recHe = rec_cool_HeII(T, CASE)*ne*xHeII;
	
	float colexH = colex_HI(T)*ne*xHI;
	
	float brem = 1.42e-27*powf(T,0.5)*ne*ne*6.242e11/n;
	
	dEdt += colH + recH + colexH;
	dEdt += colHe + recHe;
	dEdt += brem;
	
	// Adiabatic cooling:
	float H0 = 67.11*3.241e-20;
	dEdt += 3.0*H0*0.5477*powf(a, -1.5)*E;
	
	return dEdt;
}

__device__ float thin_source(float source, float fraction)
{
	if(fraction < 1.e-30)
	{
		return 0.0;
	}
	else
	{
		return source/fraction;
	}
}

__global__ void timestep(	float* rate, float* dt, float* density, float* x_N,
							float* FluxArray, float* EArray, float* background,
							int dim, float L, float a)
{
	int nelements=dim*dim*dim;
	
	int i0 = blockIdx.x*blockDim.x+threadIdx.x;
	int j0 = blockIdx.y*blockDim.y+threadIdx.y;
	int k0 = blockIdx.z;

	int ind=i0+dim*j0+dim*dim*k0;
	
	float xn[SPECIES];
	float Gamma[SPECIES];
	
	for(int nSpe=0; nSpe < SPECIES; nSpe++)
	{
		xn[nSpe] = x_N[ind + nSpe*nelements];
		Gamma[nSpe] = FluxArray[ind + nSpe*nelements];
		Gamma[nSpe] += background[nSpe];	// Ionization rate for each species (per Myr)
	}
	
	float dens = density[ind]/(a*a*a);	// Baryonic number density
	float E = EArray[ind];				// Energy (temperature) per baryon
	
	// Subcycle loop to advance the chemistry
	// First we calculate the optically thin approximation of source terms:
	float dGdx[SPECIES];
	for(int nSpe=0; nSpe<SPECIES; nSpe++)
	{
		dGdx[nSpe] = thin_source(Gamma[nSpe], xn[nSpe]);
	}
		
	// Find the max time step to advance hydrogen
	float dxdt;
	dxdt = abs(dnHIdt(xn, dGdx, dens, E));
	
	float dx = (L/DIMX)*a;
	float sig = 1.111e7; // sig[0][0]*cm in a Mpc
	float tau = max(dens*sig*dx, 3.0);
	
	// dIdt
	atomicMax(rate+1, dxdt*tau);
	
	// dnHdt
	if(tau > 0.5)
		atomicMax(rate, dxdt/xn[0]);
	dt[ind] = dxdt/xn[0];
	
	__syncthreads();
}

// This is here because derivs is inherently inline under CUDA architecture.
//#include "./inc/rkck.cu"
//#include "./inc/simpr.cu"

__global__ void ionization(	float dt, float* error, float* density, float* x_N,
							float* FluxArray, float* EArray, float* dEArray,
							float* background, int dim, float a)
{
	int nelements=dim*dim*dim;
	
	int i0 = blockIdx.x*blockDim.x+threadIdx.x;
	int j0 = blockIdx.y*blockDim.y+threadIdx.y;
	int k0 = blockIdx.z;

	int index=i0+dim*j0+dim*dim*k0;
	
	float fDtMyr = dt*3.15e13;
	
//	float t = time(1.0/a - 1.0);
	
	float xn[SPECIES];
	float xn_out[SPECIES];
	float Gamma[SPECIES];
	
	float fCumFlux = 0;
	
	for(int nSpe=0; nSpe < SPECIES; nSpe++)
	{
		xn[nSpe] = x_N[index+nSpe*nelements];
		Gamma[nSpe] = FluxArray[index+nSpe*nelements];
		Gamma[nSpe] += background[nSpe];	// Ionization rate for each species (per Myr)
		
		fCumFlux += xn[3+nSpe]*fDtMyr;
	}
	
	float dens = density[index]/(a*a*a);	// Baryonic number density
	float E = EArray[index];				// Energy (temperature) per baryon
	float dEdt = dEArray[index];
	//float dEdt = dEArray[index]+background[2]+background[3];
	
	float eps = 0.1;	// Maximum fractional change during subcycle
	float fDtRem;		// Remaining time in the subcycle loop
	
	// Subcycle loop to advance the chemistry
	// First we calculate the optically thin approximation of source terms:
	float dEdx, dGdx[SPECIES];
	for(int nSpe=0; nSpe<SPECIES; nSpe++)
	{
		dGdx[nSpe] = thin_source(Gamma[nSpe], xn[nSpe]);
	}
	dEdx = thin_source(dEdt, xn[0]);
	
	// Subcycle:
	fDtRem = fDtMyr;
	while(fDtRem > 1.0)	// One second
	{
		// Find the max time step to advance E
		float dtSubE;
		float Lam = lambda(E, xn, dens, a);
		float Heat = dEdx*xn[0];
		float dEdt = Heat - Lam;
		float rate = abs(dEdt);
		if(rate < eps*E/fDtRem)
		{
			dtSubE = fDtRem;
		}
		else
		{
			dtSubE = eps*E/rate;
		}
		
		// Find the max time step to advance hydrogen
		float dtSubH;
		rate = abs(dnHIdt(xn, dGdx, dens, E));
		if(rate < eps*0.1/fDtRem)
		{
			dtSubH = fDtRem;
		}
		else
		{
			dtSubH = eps*0.1/rate;
		}
		
		float dtSub = min(dtSubE, dtSubH);
		
		// Updating energy
		/*float E1 = E + dEdt*dtSub;
		float dEdt1 = dEdx*xn[0] - lambda(E1, xn, dens, a);
		E = MIN(2.e4*1.29e-4, E + (dEdt+dEdt1)*dtSub/2);*/
		E = E + dEdt*dtSub;
		
		step_bdf(xn_out, xn, dGdx, dens, E, dtSub);		
		for(int nSpe=0; nSpe<SPECIES; nSpe++)
		{
			if (xn_out[nSpe] < 0.0)
			{
				xn[nSpe] = 0.0;
			}
			else if (xn_out[nSpe] <= 1.0)
			{
				xn[nSpe] = xn_out[nSpe];
			}
			else
			{
				xn[nSpe] = 1.0;
			}
		}
		
		fDtRem = fDtRem - dtSub;
	}
	
	__syncthreads();
	for(int nSpe=0; nSpe<SPECIES; nSpe++)
	{
		x_N[index+nSpe*nelements] = xn[nSpe];
	}
	
	//float xerr[SPECIES];
	// Change the energy array
	if(E <= 0)
		EArray[index] = 0.0;
		//EArray[index] = E;
	else
		EArray[index] = E;
	
	/*for(int i=0; i<SPECIES; i++)
	{
		xerr[i] = xn_out[i];
	}*/
	
//	if(index == 1056832)
//		*error = xeq;
//	atomicMax(error, EArray[index]);
	atomicAdd(error, fCumFlux);
//	__syncthreads();
}

// Signum function
__device__ int sign(float x)
{
	return (x > 0) - (x < 0);
}

// Does the HEALPix math but gives a float
__device__ void fpix2vec_nest(long n, long m, float* vec)
{
	double temp[3];
	pix2vec_nest(n, m, temp);
	
	vec[0] = (float) temp[0];
	vec[1] = (float) temp[1];
	vec[2] = (float) temp[2];
}

// Takes position (x0) and direction(u) and takes a step along integer grid to x
__device__ float raystep(float* x, int* ijk, float* x0, int* ijk0, float* u)
{
	// Minimum projection, to prevent divide by 0
	float eps = 1.e-10;
	// Length of step
	float dS;
	
	// Direction of movement along each axis
	int s[3];
	for(int i=0;i<3;i++)
		s[i] = sign(u[i]);
	
	// Distance to nearest cell face along each axis
	float r[3];
	for(int i=0;i<3;i++)
	{
		if(s[i] != 0)
			r[i] = fabsf((ijk0[i] + (s[i]+1.0)/2.0) - x0[i])/MAX(eps,fabsf(u[i]));
		else
			r[i] = 1.0/eps;
	}
	
	// Initialize next step
	for(int i=0;i<3;i++)
		ijk[i] = ijk0[i];
	
	// Take the step
	if(r[0] <= r[1] && r[0] <= r[2])
	{
		dS		= r[0];
		ijk[0]	+= s[0];
	}
	if(r[1] <= r[0] && r[1] <= r[2])
	{
		dS		= r[1];
		ijk[1]	+= s[1];
	}
	if(r[2] <= r[0] && r[2] <= r[1])
	{
		dS		= r[2];
		ijk[2]	+= s[2];
	}
	
	for(int i=0;i<3;i++)
		x[i] = x0[i] + dS*u[i];
	
	return dS;
}

__device__ int rayFinish(Ray *ray, int nDom, Domain domain)
{
	if(nDom == domain.get_id())
	{
		printf("Problem: attempting to send ray to self.");
		ray->set_dom(-1);
		return 1;
	}
	
	for(int dom=0; dom<8; dom++)
	{
		if(nDom == dom)
		{
			ray->set_dom(dom);
			return 1;
		}
	}
	
	ray->set_dom(-1);
	return 1;
}

__device__ void round_down(int * I, float * X)
{
	I[0] = __double2int_rd(X[0]);
	I[1] = __double2int_rd(X[1]);
	I[2] = __double2int_rd(X[2]);
}

__device__ int BoundaryCheck(float * X, int * I, int DIM)
{
	for(int i=0; i<3; i++)
	{
		if(	X[i] < 0 || I[i] < 0 ||
			X[i] >= DIM || I[i] >= DIM)
		{
			return 1;
		}
	}
	
	return 0;
}

// For tracking rays
// X and I are ray position and gridloc, vec is the direction of the ray
// mode is for adjusting rays tracked by the tracer (0) or placed on the grid (1)
__device__ void BoundaryAdjust(float * X, int * I, float* vec, int mode, int DIM)
{
	for(int i=0; i<3; i++)
	{
		if(I[i] < 0)
		{
			X[i] += DIM;
			
			if(mode == 0)
			{
				I[i] += DIM;
			}
			else
			{
				I[i] = static_cast<int>(X[i]);
			}
		}
		
		if(I[i] >= DIM)
		{
			X[i] -= DIM;
			
			if(mode == 0)
			{
				I[i] -= DIM;
			}
			else
			{
				I[i] = static_cast<int>(X[i]);
			}
		}
	}
}

/*// For new rays
__device__ void BoundaryAdjust_new(float * X, int * I, int DIM)
{
	for(int i=0; i<3; i++)
	{
		if(X[i] < 0)
		{
			X[i] += DIM;
			I[i] = static_cast<int>(X[i]);
		}
		if(X[i] >= DIM)
		{
			X[i] -= DIM;
			I[i] = static_cast<int>(X[i]);
		}
	}
}*/

// This kernel traces rays until they split or end.
// nGrid:	number density of absorbers on the physical grid
// xGrid:	the neutral fraction of absorbers on the physical grid
// Parts:	Particles under consideration
// GamGrid:	rate of photon absorption on the physical grid
// PixList:	List of N pixels (in unique nested form)
// RayDat:	List of ray data in (R, tau_0, ..., tau_n) form
// N0:		Array of number of initial rays per particle
// Nside:	HEALPix parameter
// L:		Physical length of the side of the box
// int is used because 2e9 is enough to get to HEALPix order 13
__global__ void rayTraceKernel(	const float *nGrid, const float *xGrid,
								const source *Parts, float *GamGrid, float* dEArray,
								Ray *RayDat,  int *N, int N0,
								float L, float a, float *nfSback, Domain domain)
{
	// Determine properties of the ray to be traced:
	
	// 2+1D grid of 2D blocks. CHECKXXX
	// z dimension of grid is for particle ID
	// Blocks are 16x16 to fill the SM's in CC 3.5
	int blockID =	blockIdx.x + blockIdx.y * gridDim.x;
	int threadID = 	blockID * blockDim.x * blockDim.y
					+ threadIdx.y * blockDim.x + threadIdx.x;
	
	// Only computing Npix rays CHECKXXX
	
	if(threadID >= N0)
		return;
	
	int dim = domain.get_dim();
	int nElements = dim*dim*dim;
	int domID = domain.get_id();
	
	int xLim0[3];//, xLim1[3];
	domain.get_x0(xLim0);
	int d_ind = dim*dim*xLim0[2] + dim*xLim0[1] + xLim0[0];
	
	//domain.get_x1(xLim1);
	
	Ray *ray = RayDat + threadID;
	int partID = ray->get_part();
	int pixID, ord;
	ray->get_pix(&pixID, &ord);
	/*if(partID <0 || partID >1)
		printf("?!? %d\t%d\n", domID, partID);*/
	
	int Nside = (1 << ord);
	int Npix = 12 * Nside * Nside;
	
	// Find direction of ray
	float vec[3];
	fpix2vec_nest(Nside, pixID, vec);
	
	// Find position of the ray
	float * X;
	X = ray->position;
	
	int * I;
	I = ray->gridloc;
	
	int nDom = domain.loc(I);
	
	// Find distance to domanin wall
	int domID3[3];
	domain.get_id3(domID3);
	
	float XR[3];
	XR[0] = Parts[partID].x/dim;
	XR[1] = Parts[partID].y/dim;
	XR[2] = Parts[partID].z/dim;
	
	float r_dom = dim*raystep(XR, domID3, XR, domID3, vec);
	
	/*if(pixID < 10 && ord == 2)
		printf("%f for (%f, %f, %f)\n", r_dom, vec[0], vec[1], vec[2]);*/
	
//	printf("%d\t%e\t%e\t%e\t%e\n", pID, ray[0], X[0], X[1], X[2]);
//	printf("%d\t%e\t%d\t%d\t%d\n", pID, ray[0], I[0], I[1], I[2]);
//	printf("%d\t%e\t%e\t%e\t%e\n", pID, ray[0], vec[0], vec[1], vec[2]);
	
	// Find flux for each bin of the ray:
	float flux[FREQ_BIN_NUM];
	for(int nBin=0; nBin<FREQ_BIN_NUM; nBin++)
	{
		float source = Parts[partID].gam/Npix;
		flux[nBin] = gfn[nBin]*source*__expf(-ray->tau[nBin]);
	}
	
	// Grab the cross sections
	float sig[SPECIES][FREQ_BIN_NUM];
	sigma(sig);
	
	// Loop variables
	float X0[3], dR;
	int I0[3], ind;
	
	// Set the max distance to trace a ray
	float Rmax = 1.7320*DIMX;
	float Rsplit = sqrt(Npix/12.56636/OMEGA_RAY);
	
	float dcross = 	fabsf(Rsplit - r_dom);
	if( dcross < 2.0)
		Rsplit = Rsplit - 2.0;
	
	while(ray->R < Rsplit)
	{
		/*if(abs(X[0]-I[0]) > 2)// This is for checking boundary conditions REMOVEXXX
		if(pixID == 89829)
		{
			printf("%d %d %d %f %f %f\n", I[0], I[1], I[2], X[0], X[1], X[2]);
		}*/
		
		ind = I[0] + dim*I[1] + dim*dim*I[2] - d_ind;
		
		memcpy(I0, I, 3*sizeof(int));
		memcpy(X0, X, 3*sizeof(float));
		
		// Take a single step
		dR = raystep(X, I, X, I, vec);
		
		// Check if the ray is just outside the domain
		if(nDom != domID)
		{
			// Check if it come from the boundary
			if(PERIODIC == 1)
			{
				if(BoundaryCheck(X, I, DIMX))
				{
					BoundaryAdjust(X, I, vec, 0, DIMX);
				}
			}
			
			// Entered
			if(domain.loc(I) == domID)
			{
				ind = I[0] + dim*I[1] + dim*dim*I[2] - d_ind;
		
				memcpy(I0, I, 3*sizeof(int));
				memcpy(X0, X, 3*sizeof(float));
		
				dR += raystep(X, I, X, I, vec);
			}
			else
			{
				rayFinish(ray, nDom, domain);
				atomicSub(N, 1);
				return;
			}
		}
		
		ray->R += dR;
		
		/*if(ind < 0 || ind >= dim*dim*dim)
			printf("??? %d %d\n", domID, ind);*/
		
		// Calculate the column densities:
		float dL = (dR/DIMX)*L*a;
		
		// Hydrogen
		float nH		= nGrid[ind]*(1.0-Y_P);
		float nHI		= nH*xGrid[ind];
		float NcolHI	= 3.086e24*dL*pow(a,-3)*nHI;
		
		// Helium
		float nHe		= nGrid[ind]*0.25*Y_P;
		float nHeI		= nHe*xGrid[ind+nElements];
		float NcolHeI	= 3.086e24*dL*pow(a,-3)*nHeI;
		
		/////////	Adjacent pixel correction //////////
		float fc = 1.0;
		float Lpix = sqrtf(12.566*ray->R*ray->R/Npix);
		float Dedge = Lpix/2.0;
		int ind_c = ind;
		int del=1;
		
		float D[3];
		
		for(int i=0; i<3; i++)
		{
			float Dci;
			Dci = X0[i] + dR*vec[i]/2.0 - (I0[i]+0.5);
			Dci = Dci;
			if(abs(Dci) > 0.5)
			{
				//Dci = 1.0 - abs(Dci);
			}
			
			D[i] = Dci;
			
			float De = 0.5 - fabs(Dci);
			if(De < Dedge)
			{
				Dedge = De;
				if(Dci > 0 && I[i] + 1 < xLim0[i] + dim)//CHECKXXX
				{
					ind_c = ind + del;
				}
				else if(Dci < 0 && I[i] > xLim0[i])
				{
					ind_c = ind - del;
				}
			}
			del *= dim;
		}
		
		if(Dedge < Lpix/2.0)
			fc = powf(0.5 + Dedge/Lpix, 1.0);
		else
			fc = 1.0;
		
		//if(pixID == 10096)
		if(Dedge < 0)
		{
			fc = 1.0;
			//printf("%d: %f (%d %d %d)(%f %f %f)(%f %f %f)(%f %f %f)\n", pixID, ray->R, I0[0], I0[1], I0[2], X0[0], X0[1], X0[2], vec[0], vec[1], vec[2], D[0], D[1], D[2]);
		}
		if(D[0] > 0.5 || D[1] > 0.5 || D[2] > 0.5)
		{
			fc = 1.0;
		}
		/////////	Adjacent pixel correction //////////
		
		float gamH = 0;
		float gamHe = 0;
		float dE = 0;
		
		for(int nBin=0;nBin<FREQ_BIN_NUM;nBin++)
		{
			float dtau, dtauH, dtauHe, dampH, dampHe, absorb, A, B;
			
			// Hydrogen
			dtauH = sig[0][nBin]*NcolHI;
			dampH = exp(-dtauH);
			
			// Helium
			dtauHe = sig[1][nBin]*NcolHeI;
			dampHe = exp(-dtauHe);
			
			dtau = dtauH + dtauHe;
			
			// Keep track of total tau
			ray->tau[nBin] += dtau;
			
			// Number of absorbtions per second
			absorb = flux[nBin]*(1.0 - dampH*dampHe);
			
			// Fraction absorbed by H, He
			if(dtau < 1.e-10)
			{
				// simplify for dtau~0
				float temp_H = sig[0][nBin]*nH;
				float temp_He = sig[1][nBin]*nHe;
				A = temp_H/(temp_H+temp_He);
				B = temp_He/(temp_H+temp_He);
			}
			else
			{
				A = dtauH/dtau;
				B = dtauHe/dtau;
			}
			
			// Add total photon counts
			absorb	/= powf(L/DIMX,3)*3.086e24;	// Unit correction
			gamH	+= fc*A*absorb/nH;
			gamHe	+= fc*B*absorb/nHe;
			
			// Add the energy up CHECKXXX
			dE		+= fc*A*(energy[nBin]-13.6)*absorb/nH;
			dE		+= fc*B*(energy[nBin]-24.6)*absorb/nHe;
//			dE		+= fc*(energy[nBin]-13.6)*absorb/nGrid[ind];
			
			// Update ray luminosity
			flux[nBin] *= dampH*dampHe;
		}
		
		// Update flux array
		atomicAdd(GamGrid + ind, gamH);
		atomicAdd(GamGrid + ind + nElements, gamHe);
		
		// Update Energy array
		atomicAdd(dEArray + ind, dE);
		
		/////////	Adjacent pixel correction //////////
		float ratio = xGrid[ind_c]/xGrid[ind];
		//float ratio = 1.0;
		float gamH_c = ratio*gamH*(1.0-fc)/fc;
		float gamHe_c = ratio*gamHe*(1.0-fc)/fc;
		float dE_c = ratio*dE*(1.0-fc)/fc;
	
		atomicAdd(GamGrid + ind_c, gamH_c);
		atomicAdd(GamGrid + ind_c + nElements, gamHe_c);
		atomicAdd(dEArray + ind_c, dE_c);
		/////////	Adjacent pixel correction //////////
		
		// Apply boundary conditions, if required
		float checkX[3];
		int checkI[3];
		memcpy(checkX, ray->position, 3*sizeof(float));
		memcpy(checkI, ray->gridloc, 3*sizeof(int));
		
		float X2[3];
		int I2[3];
		memcpy(X2, ray->position, 3*sizeof(float));
		memcpy(I2, ray->gridloc, 3*sizeof(int));
		
		if(PERIODIC == 1)
		{
			
			if(BoundaryCheck(X, I, DIMX))
			{
				BoundaryAdjust(X, I, vec, 0, DIMX);
			}
			
			if(0 && pixID == 10096)
			{
				printf("A %d %d %d %f %f %f \n", I2[0], I2[1], I2[2], X2[0], X2[1], X2[2]);
				printf("B %d %d %d %f %f %f \n", I[0], I[1], I[2], X[0], X[1], X[2]);
			}
		}
		
		nDom = domain.loc(I);
		
		if(	nDom != domID )
		{
			rayFinish(ray, nDom, domain);
			atomicSub(N, 1);
			
			return;
		}
		
		// Terminate the ray above given optical depth.
		if(	ray->tau[FREQ_BIN_NUM-1] > 8.0 || ray->R > Rmax || BoundaryCheck(X, I, DIMX))
		{
			ray->set_dom(-1);
			atomicSub(N, 1);
			
			for(int nBin=0; nBin<FREQ_BIN_NUM; nBin++)
			{
				atomicAdd(nfSback + nBin, flux[nBin]); // CHECKXXX
			}
			
			return;
		}
	}
	// Add up all the rays that don't terminate
	__syncthreads();
}

// This kernel splits the rays into the next HEALPix level until they split or end.
// PixList:	List of N pixels (in unique nested form)
// RayDat:	List of ray data in (R, tau_0, ..., tau_n) form
// N0:		Number of rays
// int is used because 2e9 is enough to get to HEALPix order 13
__global__ void raySplitKernel(	Ray *RayDat_init, Ray *RayDat, int *nRays, int N0,
								Ray *RayBuf, int* nBufLoc,
								const source * source_dev, Domain domain)
{
	// 2+1D grid of 2D blocks.
	// z dimension of grid is for particle ID
	// Blocks are 16x16 to fill the SM's in CC 3.5
	int blockID =	blockIdx.x + blockIdx.y * gridDim.x;
	int threadID = 	blockID * blockDim.x * blockDim.y
					+ threadIdx.y * blockDim.x + threadIdx.x;
	
	// Only computing Npix rays
	if(threadID >= N0)
		return;
	
	Ray *ray = RayDat_init + threadID;
	
	// Terminated rays
	if(ray->get_dom() == -1)
	{
		return;
	}
	
	// Split rays
	int rayDom = ray->get_dom();
	if(rayDom == domain.get_id())
	{
		// Get a unique ID for the (first) ray
		int rayID = atomicAdd(nRays, 4);
		int partID = ray->get_part();
		int pixID, ord;
		ray->get_pix(&pixID, &ord);
		
		float origin[3];
		origin[0] = source_dev[partID].x;
		origin[1] = source_dev[partID].y;
		origin[2] = source_dev[partID].z;
	
		// Splitting into 4 rays
		for(int nSplit=0; nSplit<4; nSplit++)
		{
			Ray *ray_split = RayDat + (rayID+nSplit);
			
			int new_ID = 4*pixID + nSplit;
			int new_ord = ord + 1;
			int Nside = (1 << new_ord);
			float direction[3];
			fpix2vec_nest(Nside, new_ID, direction);
			
			ray_split->R = ray->R;
			ray_split->set_part(partID);
			ray_split->set_pix(new_ID, new_ord);
			
			ray_split->set_position(origin, ray->R, direction);
			
			// Apply boundary
			float * rayX = ray_split->position;
			int * rayI = ray_split->gridloc;
			
			float * checkX = ray->position;
			int * checkI = ray->gridloc;
			int check = -1;
			if(new_ID == check)
			{
				printf("Placing at %d %d %d %f %f %f\n", rayI[0], rayI[1], rayI[2], rayX[0], rayX[1], rayX[2]);
				printf("From %d %d %d %f %f %f\n", checkI[0], checkI[1], checkI[2], checkX[0], checkX[1], checkX[2]);
			}
			if(PERIODIC == 1)
			{
				if(BoundaryCheck(rayX, rayI, DIMX))
				{
					BoundaryAdjust(rayX, rayI, direction, 1, DIMX);
				}
			}
			if(new_ID == check)
			{
				printf("Now at %d %d %d %f %f %f\n", rayI[0], rayI[1], rayI[2], rayX[0], rayX[1], rayX[2]);
			}
			
			int splitDom = domain.loc(ray_split->gridloc);
			ray_split->set_dom(splitDom);
			
			for(int nBin=0; nBin<FREQ_BIN_NUM; nBin++)
			{
				ray_split->tau[nBin] = ray->tau[nBin];
			}
			
			// Move rays for different domains to the buffer
			if(splitDom != rayDom && splitDom != -1)
			{
				int nBufID = atomicAdd((nBufLoc + splitDom), 1);
				int pix, ord;
				ray_split->get_pix(&pix,&ord);
				RayBuf[splitDom*NUM_BUF + nBufID].copy_ray(*ray_split);
				//printf("Split ray in wrong domain! %d: %d -> %d, (%d, %d, %d) %f\n", new_ID, rayDom, splitDom, rayI[0], rayI[1], rayI[2], ray->R);
				ray_split->set_dom(-1);
			}
		}
		
		// Terminate old ray
		ray->set_dom(-1);
		return;
	}
	
	// Buffer rays
	for(int dom=0; dom<8; dom++)
	{
		if(ray->get_dom() == domain.get_id())
			continue;
		if(ray->get_dom() == dom)
		{
			// Conditional for testing REMOVEXXX
			int pixID, ord;
			ray->get_pix(&pixID, &ord);
			int Nside = (1 << ord);
			float direction[3];
			fpix2vec_nest(Nside, pixID, direction);
			//if(direction[0]<-0.75 || direction[1]<-0.75 || direction[2]<-0.75)
			if(1)
			{
				// Copy ray into buffer
				int nBufID = atomicAdd(nBufLoc + dom, 1);
				//printf("A %d\t%d\t%d\t%d\n", domain.get_id(), dom, nBufLoc[dom], dom*NUM_BUF+nBufID);
				int pix, ord;
				ray->get_pix(&pix,&ord);
			
				RayBuf[dom*NUM_BUF + nBufID].copy_ray(*ray);
				//ray->R = temp;
			}
			
			// Terminate old ray
			ray->set_dom(-1);
			return;
		}
	}
	printf("Couldn't find home for our ray!\n");
	ray->set_dom(-1);
}
